#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_add_parallel(float *a, float *b, float *c, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

__global__ void matrix_add_scalar(float *mat, float scalar, float *out, int rows, int cols) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;


  if (row < rows && col < cols) {
    int idx = row * cols + col;
    out[idx] = mat[idx] + scalar;
  }
}

__global__ void matrix_add_scalar_series(float *mat, float scalar, float *out, int rows, int cols) {
  for (int i = 0; i < rows * cols; ++i) {
    out[i] = mat[i] + scalar;
  }
}

void checkCudaError(hipError_t err, const char* msg) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

int main() {
  const int rows = 3;
  const int cols = 3;
  const int scalar = 10;

  float mat_cpp[rows * cols] = {1, 2, 3, 4, 5, 5, 7, 8, 9};

  float out_cpp[rows * cols] = {0};
  float* mat;
  float* out;

  checkCudaError(hipMalloc((void**)&mat, rows * cols * sizeof(float)), "Alloc mat");
  checkCudaError(hipMalloc((void**)&out, rows * cols * sizeof(float)), "Alloc out");

  checkCudaError(hipMemcpy(mat, mat_cpp, rows * cols * sizeof(float), hipMemcpyHostToDevice), "Memcpy mat");
  checkCudaError(hipMemcpy(out, out_cpp, rows * cols * sizeof(float), hipMemcpyHostToDevice), "Memcpy out");

  std::cout << "before run" << std::endl;

  dim3 blockSize(16, 16);
  dim3 gridSize(
      (cols + blockSize.x - 1) / blockSize.x,
      (rows + blockSize.y - 1) / blockSize.y
  );
  
  matrix_add_scalar<<<gridSize, blockSize>>>(mat, scalar, out, rows, cols);
  matrix_add_scalar_series<<<gridSize, blockSize>>>(mat, scalar, out, rows, cols);
  checkCudaError(hipGetLastError(), "Kernel launch");
  checkCudaError(hipDeviceSynchronize(), "Kernel execution");
  
  std::cout << "after run" << std::endl;

  checkCudaError(hipMemcpy(out_cpp, out, rows * cols * sizeof(float), hipMemcpyDeviceToHost), "Memcpy out back to out_cpp");

  std::cout << "[";
  for (int row = 0; row < rows; ++row) {
    std::cout << "[";
    for (int col = 0; col < cols; ++col) {
      std::cout << out_cpp[row * rows + col];
      if (col != cols-1) {
        std::cout << ", ";
      }
    }
    if (row != rows-1) {
      std::cout << "]\n";
    } else {
      std::cout << "]]\n";
    }
  }

  hipFree(mat);
  hipFree(out);
  return 0;
}