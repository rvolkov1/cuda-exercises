#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_add_series(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; ++i) {
      c[i] = a[i] + b[i];
    }
}

__global__ void vector_add_parallel(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

int main() {
    const int N = 1000000;

    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    c = (float*)malloc(sizeof(float) * N);

    for (int i = 0; i < N; ++i) {
      a[i] = 1.0f;
      b[i] = 2.0f;
    }

    checkCudaError(hipMalloc((void**)&dev_a, N * sizeof(float)), "Alloc dev_a");
    checkCudaError(hipMalloc((void**)&dev_b, N * sizeof(float)), "Alloc dev_b");
    checkCudaError(hipMalloc((void**)&dev_c, N * sizeof(float)), "Alloc dev_c");

    checkCudaError(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice), "Memcpy a");
    checkCudaError(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice), "Memcpy b");

    int num_threads = 100;

    vector_add_series<<<1, num_threads>>>(dev_a, dev_b, dev_c, N);
    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution");

    checkCudaError(hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost), "Memcpy c");

    //for (int i = 0; i < size; i++) {
    //    std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    //}

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}